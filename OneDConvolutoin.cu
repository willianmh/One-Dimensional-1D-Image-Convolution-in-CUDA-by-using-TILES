#include<stdio.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime_api.h>
#include<stdlib.h>


#define O_Tile_Width 3
#define Mask_width 3
#define width 5
#define Block_width (O_Tile_Width+(Mask_width-1))
#define Mask_radius (Mask_width/2)


__global__ void convolution_1D_tiled(float *N,float *M,float *P)
{
int index_out_x=blockIdx.x*O_Tile_Width+threadIdx.x;
int index_in_x=index_out_x-Mask_radius;
__shared__ float N_shared[Block_width];
float Pvalue=0.0;

//Load Data into shared Memory (into TILE)
if((index_in_x>=0)&&(index_in_x<width))
{
 N_shared[threadIdx.x]=N[index_in_x];
}
else
{
 N_shared[threadIdx.x]=0.0f;
}
__syncthreads();

//Calculate Convolution (Multiply TILE and Mask Arrays)
if(threadIdx.x<O_Tile_Width)
{
 //Pvalue=0.0f;
 for(int j=0;j<Mask_width;j++)
 {
  Pvalue+=M[j]*N_shared[j+threadIdx.x];
 }
 P[index_out_x]=Pvalue;
}


}

int main()
{
 float * input;
 float * Mask;
 float * output;

 float * device_input;
 float * device_Mask;
 float * device_output;

 input=(float *)malloc(sizeof(float)*width);
 Mask=(float *)malloc(sizeof(float)*Mask_width);
 output=(float *)malloc(sizeof(float)*width);

 for(int i=0;i<width;i++)
 {
  input[i]=1.0;
 }

 for(int i=0;i<Mask_width;i++)
 {
  Mask[i]=1.0;
 }
  printf("\nInput:\n");
  for(int i=0;i<width;i++)
  {
   printf(" %0.2f\t",*(input+i));
  }
  printf("\nMask:\n");
   for(int i=0;i<Mask_width;i++)
   {
    printf(" %0.2f\t",*(Mask+i));
   }

 hipMalloc((void **)&device_input,sizeof(float)*width);
 hipMalloc((void **)&device_Mask,sizeof(float)*Mask_width);
 hipMalloc((void **)&device_output,sizeof(float)*width);

 hipMemcpy(device_input,input,sizeof(float)*width,hipMemcpyHostToDevice);
 hipMemcpy(device_Mask,Mask,sizeof(float)*Mask_width,hipMemcpyHostToDevice);

 dim3 dimBlock(Block_width,1,1);
 dim3 dimGrid((((width-1)/O_Tile_Width)+1),1,1);
 convolution_1D_tiled<<<dimGrid,dimBlock>>>(device_input,device_Mask,device_output);

 hipMemcpy(output,device_output,sizeof(float)*width,hipMemcpyDeviceToHost);

 printf("\nOutput:\n");
 for(int i=0;i<width;i++)
 {
  printf(" %0.2f\t",*(output+i));
 }

 hipFree(device_input);
 hipFree(device_Mask);
 hipFree(device_output);
 free(input);
 free(Mask);
 free(output);

printf("\n\nNumber of Blocks: %d ",dimGrid.x);
printf("\n\nNumber of Threads Per Block: %d ",dimBlock.x);

return 0;
}
